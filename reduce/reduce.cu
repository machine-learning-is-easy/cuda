#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void global_memory_reduce_kernel(float * d_out, float *d_in)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	// reduction process
	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			d_in[myId] += d_in[myId + s];
		}

		__syncthreads(); // need to synchonize all thread here
	}

	if (tid == 0)
	{
		d_out[blockIdx.x] = d_in[myId];
	}

}

__global__ void share_memory_reduce_kernel(float * d_out, float * d_in)
{
	extern __shared__ float sdata[];

	int myId = threadIdx.x + blockDim.x * blockIdx.x;
	int tid = threadIdx.x;

	// load the shared memory from global memory
	sdata[tid] = d_in[myId];
	__syncthreads();  // synchnize all thread

	// reduction
	for (unsigned int s = blockDim.x / 2; s>0; s >>=1)
	{
		if (tid < s)
		{
			sdata[tid] += sdata[tid + s];
		}
		__syncthreads(); // syncthread();
	}

	if (tid == 0)
	{
		d_out[blockIdx.x] = sdata[0];
	}
}

void reduce(float * d_out, float * d_intermediate, float * d_in, int size, bool useSharedMemory)
{
	const int maxThreadsPerBlock = 1024;
	int threads = maxThreadsPerBlock;
	int blocks = size / maxThreadsPerBlock;

	if (useSharedMemory)
	{
		share_memory_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_intermediate, d_in);
	}
	else
	{
		global_memory_reduce_kernel<<<blocks, threads>>>(d_intermediate, d_in);
	}

	threads = blocks;
	blocks = 1;

	if (useSharedMemory)
	{
		share_memory_reduce_kernel<<<blocks, threads, threads * sizeof(float)>>>(d_out, d_intermediate);
	}
	else
	{
		global_memory_reduce_kernel<<<blocks, threads>>>(d_out, d_intermediate);
	}
}


int main(int argc, char **argv)
{
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0)
	{
		printf("error, no devices supporting CUDA. \n");
		exit(EXIT_FAILURE);
	}

	int dev = 0;
	hipSetDevice(dev);

	hipDeviceProp_t devProps;
	if (hipGetDeviceProperties(&devProps, dev) == 0)
	{
		printf("Using device %d:\n", dev);
		printf("%s; global memory: % db; compute v%d.%d; clock: %d kHz\n",
				devProps.name, (int)devProps.totalGlobalMem,
				(int) devProps.major, (int)devProps.minor,
				(int)devProps.clockRate);
	}

	const int ARRAY_SIZE = 1 << 20;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	float h_in[ARRAY_SIZE];
	float sum = 0.0f;
	for (int i=0; i < ARRAY_SIZE; i++)
	{
		//initialization of the array element
		h_in[i] = -1.0f +  float(random()/float(RAND_MAX/2.0f));
		sum += h_in[i];
	}

	float * d_in, * d_intermediate, * d_out;

	//alocate GPU memory
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_intermediate, ARRAY_BYTES);
	hipMalloc((void **) &d_out, sizeof(float));

	// transfer the input array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
	int whichKernel = 0;
	if (argc == 2)
	{
		whichKernel = atoi(argv[1]);
		printf("Input parameter is %d\n", whichKernel);
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf("Input parameter is %d \n", whichKernel);
	switch(whichKernel){
		case 0:
			printf("Running global reduce \n");
			hipEventRecord(start, 0);
			for (int i=0; i < 100; i++)
			{     
				reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, false);
			}
			hipEventRecord(stop, 0);
			break;
		case 1:
			printf("Runing reduce with shared memory \n");
			hipEventRecord(start, 0);
			for (int i = 0; i < 100; i ++)
			{
				reduce(d_out, d_intermediate, d_in, ARRAY_SIZE, true);
			}
			hipEventRecord(stop, 0);
			break;
		default:
			printf("error: Non kernel is available\n");
			exit(EXIT_FAILURE);

	}
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	elapsedTime /= 100.0f;

	float h_out;
	hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);
	printf("average time elapsed: %f\n", elapsedTime);

	//free GPU memory allocation
	hipFree(d_in);
	hipFree(d_intermediate);
	hipFree(d_out);
	return 0;
}
